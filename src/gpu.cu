#include "gpu.h"

//Got directly from "Accelerated Ray Tracing in One Weekend in CUDA" tutorial
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

void allocateGPUSharedMem(void ** ptr, int size){
    checkCudaErrors(hipMallocManaged((void **)ptr, size));
}

void allocateGPUMem(void ** ptr, int size){
    checkCudaErrors(hipMalloc((void **)ptr  , size));
}

void freeGPUMem(void* ptr){
    checkCudaErrors(hipFree(ptr));
}

void syncGPU(){
    checkCudaErrors(hipDeviceSynchronize());
}

void checkGPUErrors(){
    checkCudaErrors(hipGetLastError());
}
