#include "hip/hip_runtime.h"
#include "render.h"

__device__
vec4_t raycast (ray_t ray, Object ** obj_list, Light ** light_list, int n_obj, int n_light, float near, int depth){
    float dist = INFINITY;
    collision_t closest_collision;
    vec4_t accucolor = vec4_t{ 0.8, 0.8, 0.8, 0.0 }; //white room
    //vec4_t accucolor = vec4_t{ 0.0, 0.57, 0.87, 0.0 }; //blue room
    //vec4_t accucolor = vec4_t{ 0.5, 0.5, 0.5, 0.0 }; //Medium room
    Object *closest_obj = nullptr;

    float curr_near = near;
    ray_t curr_ray= ray;

    for (int i = 0; i < n_obj; ++i)
    {
        collision_t col = obj_list[i]->geometry->CollideClosest(curr_ray);//Collision
        float o_dist = col.distance;
        
        if (o_dist < dist && o_dist > curr_near) {
            dist = o_dist;
            closest_collision = col;
            closest_obj = obj_list[i];
        }
    }

    if (closest_obj){
        Material mat = *(closest_obj->material);
        vec4_t normal = closest_obj->geometry->getNormal(closest_collision.pint);
        accucolor = mat.ka;
        
        for (int i = 0; i < n_light; ++i){
    
            vec4_t direction = light_list[i]->GetLightDirection(closest_collision.pint);
            float dot_d_n = dot(direction, normal);
            
            if (dot_d_n >= 0){
                accucolor += (light_list[i]->Illumination(mat, normal, curr_ray.direction * (-1.0), closest_collision.pint)*mat.alfa);
            }
        }
    
        
        if (mat.reflection && depth < 3) {
            const vec4_t reflect = (reflection(ray.direction * (-1.0), normal)).unit();
            accucolor += raycast(ray_t{ closest_collision.pint, reflect}, obj_list, light_list, n_obj, n_light, 0.00001, depth+1) * mat.reflection;
            accucolor = accucolor * 0.5; //HACK
        }

        if (mat.alfa < 1 && depth < 3) {
            vec4_t vec_offset = closest_collision.pint + ray.direction * 0.001; //Avoid bugs
            const vec4_t refract_ray = refract(ray.direction, normal, mat.ior, 1);
            accucolor += raycast(ray_t{ vec_offset , refract_ray }, obj_list, light_list, n_obj, n_light, 0.00001, depth + 1)*(1-mat.alfa);
            accucolor = accucolor * 0.5; //HACK
        }



    }
    return accucolor;
}

__global__
void render (camera_t * cam, Object ** obj_list, Light ** light_list, int dn_obj, int dn_light, vec4_t * buffer) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    const vec4_t base = (cam->lookat - cam->position).unit();
    const vec4_t up = cam->up * (cam->dimensions.y / cam->resolution.y);
    const vec4_t left = cross(cam->up, base) * (cam->dimensions.x / cam->resolution.x);

    const int width  = (int)cam->resolution.x;
    const int height = (int)cam->resolution.y;

    const vec4_t origin_screen = base * cam->n + up * (height*(-0.5) + 0.5) + left * (width*(0.5) - 0.5);
    
    if((x >= width) || (y >= height)) return;

    //Scan screen
    vec4_t direction = (origin_screen + up * (y) + left * (-x)).unit();
    buffer[y*width + x] = raycast(ray_t{cam->position, direction}, obj_list, light_list, dn_obj, dn_light, cam->n, 0);
}

__global__
void create_world(Object ** obj_list, Light ** light_list){

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(obj_list)     = new Object((char *)"Bronze Sphere", new Material((char *)"Bronze", vec4_t{ 0.2125, 0.1275, 0.054, 0.0 }, vec4_t{ 0.714, 0.4284, 0.18144, 0.0 }, vec4_t{ 0.393548, 0.271906, 0.166721, 0.0 }, 25.6), new Sphere(vec4_t{1.0,0.0,0.0,0.0}, 0.8));
        *(obj_list+1)   = new Object((char *)"Gold Sphere", new Material((char *)"Gold", vec4_t{ 0.24725, 0.1995, 0.0745, 0.0 }, vec4_t{ 0.75164, 0.60648, 0.22648, 0.0 }, vec4_t{ 0.628281, 0.555802, 0.366065, 0.0 }, 51.2), new Sphere(vec4_t{-1.0,0.0,0.0,0.0}, 0.8));
        *(obj_list+2)   = new Object((char *)"Mirror Sphere", new Material((char *)"Mirror", vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 1.0, 1.0, 1.0, 0 }, 200.0, 1.0, 0.8), new Sphere(vec4_t{0.0,0.0,-2.0,0.0}, 0.8));
        *(obj_list+3)   = new Object((char *)"Refract Sphere", new Material((char *)"Refract", vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 0.2, 0.2, 0.2, 0 }, 2, 0.5, 0.5, 1.1), new Sphere(vec4_t{0.0,0.0,2.0,0.0}, 0.8));
    
        *(light_list  )   = new DirectionalLight(vec4_t{ 1.0, 1.0, 1.0, 0.0 }, vec4_t{ 1.0, 1.0, 1.0, 0.0 });
        *(light_list+1)   = new DirectionalLight(vec4_t{ 1.0, 1.0, 1.0, 0.0 }, vec4_t{ -1.0, 1.0, 1.0, 0.0 });
        *(light_list+2)   = new DirectionalLight(vec4_t{ 1.0, 1.0, 1.0, 0.0 }, vec4_t{ 1.0, -1.0, 1.0, 0.0 });
    }
}

Render::Render(int _nx, int _ny){
    n_obj = 4;
    n_light = 3;

    nx = _nx;
    ny = _ny;
}

Render::~Render(){
    //
}

void Render::initWord(){
    //Allocate Camera
    allocateGPUSharedMem((void **)&cam, sizeof(camera_t));
    *cam = {
        vec4_t{0.0,3.0,10.0,1.0}, //Position
        vec4_t{0.0,0.0,0.0,1.0}, //Lookat
        vec4_t{0.0,1.0,0.0,0.0}, //Up
        vec2_t{(float)nx,(float)ny}, // Resolution
        vec2_t{2.0,2.0}, // Dimension
        2.0 // Near

    };


    //Allocate buffer
    int buffer_space = (int) cam->resolution.x * cam->resolution.y * sizeof(vec4_t);
    allocateGPUSharedMem((void **)&h_buffer, buffer_space);
    //allocateGPUMem((void **)&h_buffer , buffer_space);

    //Allocate scene elements
    allocateGPUMem((void **)&d_obj_list  , sizeof(Object *) * n_obj);
    allocateGPUMem((void **)&d_light_list, sizeof(Light  *) * n_light);

    create_world<<<1,1>>>(d_obj_list, d_light_list);
}

void Render::renderScene (int nx_block, int ny_block, int nx_thread, int ny_thread, vec4_t * buff ){
    //Block and Thread
    dim3 n_blocks (nx_block, ny_block);
    dim3 n_threads(nx_thread, ny_thread);
    //vec4_t a = h_buffer[0];
    
    render<<<n_blocks, n_threads>>>(cam, d_obj_list, d_light_list, n_obj, n_light, h_buffer);
    checkGPUErrors();
    syncGPU();
    //buff = h_buffer;
    hipMemcpy(buff, h_buffer, (int) nx * ny * sizeof(vec4_t), hipMemcpyDeviceToHost);
}

void Render::updateCamera (float desloc, float deltaX, float deltaY){
    /*
    *cam = {
        vec4_t{0.0,3.0,10.0,1.0}, //Position
        vec4_t{0.0,0.0,0.0,1.0}, //Lookat
        vec4_t{0.0,1.0,0.0,0.0}, //Up
        vec2_t{(float)nx,(float)ny}, // Resolution
        vec2_t{2.0,2.0}, // Dimension
        2.0 // Near

    };
    */

    //cam->position = cam->position + (cam->position - cam->lookat)*(desloc*0.01f);//Desloc

    //Rotate
    if (deltaX != 0.0 && deltaY != 0.0){
        /* code */
        vec4_t v = cross(cam->lookat - cam->position, cam->up);
        //cam->Transform(rotateArbitrary(-deltaY, v), cam->lookat);
        //cam->Transform(y_rotate(deltaX), cam->lookat);
        cam->Transform(y_rotate(deltaX));
    }


}

void Render::getBuffer(vec4_t * color_buffer){
    //cout << "Test" << endl;
    

    //color_buffer[1] = vec4_t{0,0,0,0};

    /*
    color_buffer = h_buffer;
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*h_buffer[pixel_index].r);
            int ig = int(255.99*h_buffer[pixel_index].g);
            int ib = int(255.99*h_buffer[pixel_index].b);
            if (ir < 0) ir = 0;
            if (ig < 0) ig = 0;
            if (ib < 0) ib = 0;
            color_buffer[pixel_index].r = ir;
            color_buffer[pixel_index].g = ig;
            color_buffer[pixel_index].b = ib;
            //resultfile << ir << " " << ig << " " << ib << "\n";
        }
    }*/
}