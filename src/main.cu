#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <time.h>
#include <fstream>

#include "Geometry.h"
#include "Sphere.h"
#include "Material.h"
#include "Object.h"
#include "Light.h"
#include "AmbientLight.h"
#include "DirectionalLight.h"

#include "vec4.h"
#include "ray.h"
#include "camera.h"

using namespace std;

//Got directly from "Accelerated Ray Tracing in One Weekend in CUDA" tutorial
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec4_t raycast (ray_t ray, Object ** obj_list, Light ** light_list, int n_obj, int n_light, float near, int depth);
__global__ void render (camera_t * cam, Object ** obj_list, Light ** light_list, int n_obj, int n_light, vec4_t * buffer);
__global__ void create_world(Object ** obj_list, Light ** light_list);


int main(int argc, char const *argv[])
{
    //REsult File
    ofstream resultfile ("result.ppm");
    //Dimensions
    int nx = 512;
    int ny = 512;
    //int ns = 10;
    //Thread Size
    int tx = 8;
    int ty = 8;
    //Block and Thread
    dim3 n_blocks (nx+1/tx, ny+1/ty);
    dim3 n_threads(tx, ty);

    vec4_t * buffer; //Color buffer
    
    //Scene list
    Object ** d_obj_list;
    Light  ** d_light_list;

    //Camera
    camera_t * cam;

    //Allocate Camera
    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(camera_t)));
    *cam = {
        vec4_t{0.0,3.0,10.0,1.0}, //Position
        vec4_t{0.0,0.0,0.0,1.0}, //Lookat
        vec4_t{0.0,1.0,0.0,0.0}, //Up
        vec2_t{(float)nx,(float)ny}, // Resolution
        vec2_t{2.0,2.0}, // Dimension
        2.0 // Near

    };


    //Allocate buffer
    float bufer_space = cam->resolution.x * cam->resolution.y * sizeof(vec4_t);
    checkCudaErrors(hipMallocManaged((void **)&buffer, bufer_space));

    //Allocate scene elements
    checkCudaErrors(hipMalloc((void **)&d_obj_list  , sizeof(Object *) * 4));
    checkCudaErrors(hipMalloc((void **)&d_light_list, sizeof(Light  *) * 3));
    create_world<<<1,1>>>(d_obj_list, d_light_list);

    //Render
    clock_t begin = clock();
    render<<<n_blocks, n_threads>>>(cam, d_obj_list, d_light_list, 4, 3, buffer);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    cout << "It spent: " << time_spent <<"s" << " to render." << endl;

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Generate PPM Image
    resultfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*buffer[pixel_index].r);
            int ig = int(255.99*buffer[pixel_index].g);
            int ib = int(255.99*buffer[pixel_index].b);
            if (ir < 0) ir = 0;
            if (ig < 0) ig = 0;
            if (ib < 0) ib = 0;
            resultfile << ir << " " << ig << " " << ib << "\n";
        }
    }


    return 0;
}

__device__
vec4_t raycast (ray_t ray, Object ** obj_list, Light ** light_list, int n_obj, int n_light, float near, int depth){
    float dist = INFINITY;
    collision_t closest_collision;
    vec4_t accucolor = vec4_t{ 0.8, 0.8, 0.8, 0.0 }; //white room
    //vec4_t accucolor = vec4_t{ 0.0, 0.57, 0.87, 0.0 }; //blue room
    //vec4_t accucolor = vec4_t{ 0.5, 0.5, 0.5, 0.0 }; //Medium room
    Object *closest_obj = nullptr;

    float curr_near = near;
    ray_t curr_ray= ray;

    for (int i = 0; i < n_obj; ++i)
    {
        collision_t col = obj_list[i]->geometry->CollideClosest(curr_ray);//Collision
        float o_dist = col.distance;
        
        if (o_dist < dist && o_dist > curr_near) {
            dist = o_dist;
            closest_collision = col;
            closest_obj = obj_list[i];
        }
    }

    if (closest_obj){
        Material mat = *(closest_obj->material);
        vec4_t normal = closest_obj->geometry->getNormal(closest_collision.pint);
        accucolor = mat.ka;
        
        for (int i = 0; i < n_light; ++i){
    
            vec4_t direction = light_list[i]->GetLightDirection(closest_collision.pint);
            float dot_d_n = dot(direction, normal);
            
            if (dot_d_n >= 0){
                accucolor += (light_list[i]->Illumination(mat, normal, curr_ray.direction * (-1.0), closest_collision.pint)*mat.alfa);
            }
        }
    
        
        if (mat.reflection && depth < 3) {
            const vec4_t reflect = (reflection(ray.direction * (-1.0), normal)).unit();
            accucolor += raycast(ray_t{ closest_collision.pint, reflect}, obj_list, light_list, n_obj, n_light, 0.00001, depth+1) * mat.reflection;
            accucolor = accucolor * 0.5; //HACK
        }

        if (mat.alfa < 1 && depth < 3) {
            vec4_t vec_offset = closest_collision.pint + ray.direction * 0.001; //Avoid bugs
            const vec4_t refract_ray = refract(ray.direction, normal, mat.ior, 1);
            accucolor += raycast(ray_t{ vec_offset , refract_ray }, obj_list, light_list, n_obj, n_light, 0.00001, depth + 1)*(1-mat.alfa);
            accucolor = accucolor * 0.5; //HACK
        }



    }
    return accucolor;
}

__device__
vec4_t test_func(){
    return vec4_t{1.0,1.0,1.0,1.0};
}

__global__
void render (camera_t * cam, Object ** obj_list, Light ** light_list, int n_obj, int n_light, vec4_t * buffer) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    const vec4_t base = (cam->lookat - cam->position).unit();
    const vec4_t up = cam->up * (cam->dimensions.y / cam->resolution.y);
    const vec4_t left = cross(cam->up, base) * (cam->dimensions.x / cam->resolution.x);

    const int width  = (int)cam->resolution.x;
    const int height = (int)cam->resolution.y;

    const vec4_t origin_screen = base * cam->n + up * (height*(-0.5) + 0.5) + left * (width*(0.5) - 0.5);
    
    if((x >= width) || (y >= height)) return;

    //Scan screen
    vec4_t direction = (origin_screen + up * (y) + left * (-x)).unit();
    buffer[y*width + x] = raycast(ray_t{cam->position, direction}, obj_list, light_list, n_obj, n_light, cam->n, 0);


}

__global__
void create_world(Object ** obj_list, Light ** light_list){

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(obj_list)     = new Object((char *)"Bronze Sphere", new Material((char *)"Bronze", vec4_t{ 0.2125, 0.1275, 0.054, 0.0 }, vec4_t{ 0.714, 0.4284, 0.18144, 0.0 }, vec4_t{ 0.393548, 0.271906, 0.166721, 0.0 }, 25.6), new Sphere(vec4_t{1.0,0.0,0.0,0.0}, 0.8));
        *(obj_list+1)   = new Object((char *)"Gold Sphere", new Material((char *)"Gold", vec4_t{ 0.24725, 0.1995, 0.0745, 0.0 }, vec4_t{ 0.75164, 0.60648, 0.22648, 0.0 }, vec4_t{ 0.628281, 0.555802, 0.366065, 0.0 }, 51.2), new Sphere(vec4_t{-1.0,0.0,0.0,0.0}, 0.8));
        *(obj_list+2)   = new Object((char *)"Mirror Sphere", new Material((char *)"Mirror", vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 1.0, 1.0, 1.0, 0 }, 200.0, 1.0, 0.8), new Sphere(vec4_t{0.0,0.0,-2.0,0.0}, 0.8));
        *(obj_list+3)   = new Object((char *)"Refract Sphere", new Material((char *)"Refract", vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 1.0, 1.0, 1.0, 0 }, vec4_t{ 0.2, 0.2, 0.2, 0 }, 2, 0.5, 0.5, 1.1), new Sphere(vec4_t{0.0,0.0,2.0,0.0}, 0.8));
    
        *(light_list  )   = new DirectionalLight(vec4_t{ 1.0, 1.0, 1.0, 0.0 }, vec4_t{ 1.0, 1.0, 1.0, 0.0 });
        *(light_list+1)   = new DirectionalLight(vec4_t{ 1.0, 1.0, 1.0, 0.0 }, vec4_t{ -1.0, 1.0, 1.0, 0.0 });
        *(light_list+2)   = new DirectionalLight(vec4_t{ 1.0, 1.0, 1.0, 0.0 }, vec4_t{ 1.0, -1.0, 1.0, 0.0 });
    }


}
