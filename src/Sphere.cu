#include "hip/hip_runtime.h"
#include "Sphere.h"

CUDA_DEV
Sphere::Sphere()
{

}

CUDA_DEV
Sphere::Sphere(const vec4_t &C, float R){
    center = C;
    radius = R;
}

CUDA_DEV
Sphere::~Sphere()
{

}

//TODO: Get the Intercept Point
CUDA_DEV
collision_t Sphere::CollideClosest(const ray_t &ray)
{
    collision_t col;
    col.geometry = this;

    const vec4_t &W = ray.origin - center;     // The difference of P0 - C

    // Coefficients of the equation
    //const float A = 1.0f;//considering that the direction is already normalized
    const float B = dot(W, ray.direction);
    const float C = dot(W, W) - (radius * radius);
    // Discriminant
    const float Delta = (B*B - C);

    if (Delta < 0.0f) { col.distance = INFINITY; return col; }

    //Distance
    col.distance = -(B + sqrtf(Delta));

    //Intersection Point
    col.pint = ray.origin + (ray.direction * col.distance); // Surface 

    return col;
}

CUDA_DEV
vec4_t Sphere::getNormal(const vec4_t &point)
{
    vec4_t n = point - center;
    n.unit();
    return n;
}