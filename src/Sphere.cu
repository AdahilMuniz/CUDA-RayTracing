#include "hip/hip_runtime.h"
#include "Sphere.h"

__device__
Sphere::Sphere()
{

}

__device__
Sphere::Sphere(const vec4_t &C, float R){
    center = C;
    radius = R;
}

__device__
Sphere::~Sphere()
{

}

//TODO: Get the Intercept Point
__device__
collision_t Sphere::CollideClosest(const ray_t &ray)
{
    collision_t col;
    col.geometry = this;

    const vec4_t &W = ray.origin - center;     // The difference of P0 - C

    // Coefficients of the equation
    //const float A = 1.0f;//considering that the direction is already normalized
    const float B = dot(W, ray.direction);
    const float C = dot(W, W) - (radius * radius);
    // Discriminant
    const float Delta = (B*B - C);

    if (Delta < 0.0f) { col.distance = INFINITY; return col; }

    //Distance
    col.distance = -(B + sqrtf(Delta));

    //Intersection Point
    col.pint = ray.origin + (ray.direction * col.distance); // Surface 

    return col;
}

__device__
vec4_t Sphere::getNormal(const vec4_t &point)
{
    vec4_t n = point - center;
    n.unit();
    return n;
}